#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
// CUDA and CUBLAS functions
#include ""
#include "helper_string.h"
#include "hip/hip_runtime_api.h"
#include "mex.h"
#include <stdio.h>
#include <hip/hip_runtime.h>



#ifndef min
#define min(a,b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif

typedef struct _matrixSize      // Optional Command-line multiplier for matrix sizes
{
	unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
} sMatrixSize;


bool initCuda(hipDeviceProp_t &deviceProp, size_t &avail, size_t needed)
{
	// Initialize CUDA
	int devID;
	hipError_t error;
	error = hipGetDevice(&devID);
	if (error != hipSuccess)
	{
		mexPrintf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
		return false;
	}
	error = hipGetDeviceProperties(&deviceProp, devID);
	if (error != hipSuccess)
	{
		mexPrintf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
		return false;
	}

	size_t total;
	error = hipMemGetInfo(&avail, &total);
	if (error != hipSuccess)
		mexPrintf("GPU Device %d: \"%s\" with compute capability %d.%d, %.0f/%.0f MB, need %.0f MB \n", devID, deviceProp.name, deviceProp.major, deviceProp.minor, (double)(avail) / 1e6, (double)(total) / 1e6, (double)needed / 1e6);

	return true;
}

void mexFunction(int nlhs, mxArray *plhs[],
	int nrhs, const mxArray *prhs[]) {
	hipError_t cudaStatus;
	sMatrixSize matrix_size;
	hipDeviceProp_t deviceProp;

	if (nrhs < 2 || nlhs != 1)
	{
		mexPrintf("Use: [C] = CudaFastMult(A,B);");
		return;
	}


	float *phaseBasis = (float*)mxGetData(prhs[0]);
	float *K = (float*)mxGetData(prhs[1]);
	const size_t *dimA = mxGetDimensions(prhs[0]);
	const size_t *dimB = mxGetDimensions(prhs[1]);

	if (!mxIsSingle(prhs[0]) || !mxIsSingle(prhs[1]))
	{
		mexPrintf("Currently supporting only single class variables\n");
		return;
	}

	matrix_size.uiWA = dimA[1];
	matrix_size.uiHA = dimA[0];
	matrix_size.uiWB = dimB[1];
	matrix_size.uiHB = dimB[0];
	matrix_size.uiWC = matrix_size.uiWB;
	matrix_size.uiHC = matrix_size.uiHA;

	plhs[0] = mxCreateNumericMatrix(dimA[0], dimB[1], mxSINGLE_CLASS, mxREAL);
	float *h_A = (float *)mxGetData(prhs[0]);
	float *h_B = (float *)mxGetData(prhs[1]);
	float *h_C = (float *)mxGetData(plhs[0]);

	size_t size_A = matrix_size.uiWA * matrix_size.uiHA;
	size_t mem_size_A = sizeof(float) * size_A;
	size_t size_B = matrix_size.uiWB * matrix_size.uiHB;
	size_t mem_size_B = sizeof(float) * size_B;
	size_t size_C = matrix_size.uiWC * matrix_size.uiHC;
	size_t mem_size_C = sizeof(float) * size_C;

	size_t availMemory;
	if (!initCuda(deviceProp, availMemory, mem_size_A + mem_size_B + mem_size_C))
		return;

	// matrix B is too big. Splitting it column wise.
	
	int maximumColumnsInMemory = floor(0.4 * (availMemory - mem_size_A) / (matrix_size.uiHC + matrix_size.uiHB) / sizeof(float));
		
		//floor(0.4*(availMemory - mem_size_A) / 2.0 / sizeof(float) / matrix_size.uiHC);


	size_t memorychunk_B = min(maximumColumnsInMemory * matrix_size.uiHB * sizeof(float), mem_size_B);
	size_t memorychunk_C = min(maximumColumnsInMemory * matrix_size.uiHC * sizeof(float), mem_size_C);
	int numIterations = ceil((float)matrix_size.uiWC / maximumColumnsInMemory);
	float *d_A, *d_B, *d_C;
	checkCudaErrors(hipMalloc((void **)&d_A, mem_size_A));
	checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
	
	// A Always stays in memory

	checkCudaErrors(hipMalloc((void **)&d_C, memorychunk_C));
	checkCudaErrors(hipMalloc((void **)&d_B, memorychunk_B));

	hipblasHandle_t handle;
	checkCudaErrors(hipblasCreate(&handle));

	for (int iteration = 0; iteration < numIterations; iteration++)
	{

		int StartColumn = iteration*maximumColumnsInMemory;
		int EndColumn = min((iteration + 1)*maximumColumnsInMemory - 1, matrix_size.uiWC - 1);
		int numColumnstoCompute = EndColumn - StartColumn + 1;
		//mexPrintf("Iteration %d/%d, Columns [%d-%d] of %d\n", 1+iteration, numIterations, StartColumn, EndColumn, matrix_size.uiWC - 1);
		size_t mem_cropped_B = numColumnstoCompute * matrix_size.uiHB * sizeof(float);
		size_t mem_offset_B = matrix_size.uiHB *StartColumn;
		// Load sub-matrix of B into gpu memory

		checkCudaErrors(hipMemcpy(d_B, h_B + mem_offset_B, mem_cropped_B, hipMemcpyHostToDevice));

		const float alpha = 1.0f;
		const float beta = 0.0f;
		checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiHA, numColumnstoCompute,
			matrix_size.uiWA, &alpha, d_A, matrix_size.uiHA, d_B, matrix_size.uiHB, &beta, d_C, matrix_size.uiHC));

		size_t mem_cropped_C = numColumnstoCompute * matrix_size.uiHC * sizeof(float);
		size_t mem_offset_C = matrix_size.uiHC *StartColumn;


		checkCudaErrors(hipDeviceSynchronize());
		// copy result from device to host
		checkCudaErrors(hipMemcpy(h_C + mem_offset_C, d_C, mem_cropped_C, hipMemcpyDeviceToHost));
	}
	checkCudaErrors(hipFree(d_A));
	checkCudaErrors(hipFree(d_B));
	checkCudaErrors(hipFree(d_C));
	// Destroy the handle
	checkCudaErrors(hipblasDestroy(handle));

	hipDeviceReset();



	return;
}
