#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
// CUDA and CUBLAS functions
#include ""
#include <hip/hip_math_constants.h>
#include "helper_string.h"
#include "hip/hip_runtime_api.h"
#include <mex.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


#ifndef min
#define min(a,b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif


#define numColumnsPerFrame 128
#define numRowsPerFrame  768
#define stride 128
#define frameSizeInBytes (size_t(128) * 768)


__global__ void LeeKernel3(float *phases, unsigned char *out, int numReferencePixels, int leeBlockSize, float* carrierFreq, int patternSizeX, int patternSizeY, float *rot)
{
	size_t column_global = blockIdx.x*blockDim.x + threadIdx.x;
	long y = blockIdx.y*blockDim.y + threadIdx.y;
	long x = column_global % numColumnsPerFrame;
	//size_t z_global = inputOffsetPlane + (column_global / numColumnsPerFrame);
	size_t z_local = column_global / numColumnsPerFrame;

	float alpha[8] = { 0, 0, 0, 0, 0, 0, 0, 0 };
	unsigned char B[8];

	
	if ((y >= numReferencePixels) && (y < numRowsPerFrame - numReferencePixels) && (x * 8 >= numReferencePixels) && (x * 8 < 768 - numReferencePixels))
	{
		// query inputs!
		int sampleY = (y - numReferencePixels) / leeBlockSize;
		for (int k = 0; k<8; k++)
		{
			int sampleX = (8 * x - numReferencePixels + k) / leeBlockSize;
			alpha[k] = phases[z_local*patternSizeY*patternSizeX + sampleX*patternSizeY + sampleY];
		}
	}

			for (int k = 0; k<8; k++)
		{
			//float carrierWave = (x * 8 + k) - y; // old version rotation
			float carrierWave = cos(rot[z_local])*(x * 8 + k) + sin(rot[z_local])*y; // carrier wave rotation
			B[k] = (0.5 * (1 + cos(2.0f * (float)HIP_PI_F*(carrierWave)* (carrierFreq[z_local])-alpha[k]))) > 0.5;
		}

	
	out[frameSizeInBytes*z_local + y * stride + x] = B[0] * 128 | B[1] * 64 | B[2] * 32 | B[3] * 16 | B[4] * 8 | B[5] * 4 | B[6] * 2 | B[7] * 1;
}




bool initCuda(hipDeviceProp_t &deviceProp, size_t &avail, size_t needed)
{
	// Initialize CUDA
	int devID;
	hipError_t error;
	error = hipGetDevice(&devID);
	if (error != hipSuccess)
	{
		mexPrintf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
		return false;
	}
	error = hipGetDeviceProperties(&deviceProp, devID);
	if (error != hipSuccess)
	{
		mexPrintf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
		return false;
	}


	size_t total;
	error = hipMemGetInfo(&avail, &total);
	if (error != hipSuccess)
		printf("GPU Device %d: \"%s\" with compute capability %d.%d, %.0f/%.0f MB, allocating %.0f \n", devID, deviceProp.name, deviceProp.major, deviceProp.minor, (double)(avail) / 1e6, (double)(total) / 1e6, (double)needed/1e6);

	return true;
}

void mexFunction(int nlhs, mxArray *plhs[],
	int nrhs, const mxArray *prhs[]) {
	hipError_t cudaStatus;
	hipDeviceProp_t deviceProp;


	if (nrhs < 5 || nlhs != 1)
	{
		mexPrintf("Use: [Output:768x128xN] = CudaLeeHologram(Inputs [MxMxN], numReferencePixels, leeBlockSize, carrierFreq, rotation);");
		return;
	}

	
	float *phases  = (float*)mxGetData(prhs[0]);
	int numReferencePixels = *(double*)mxGetData(prhs[1]);
	int leeBlockSize = *(double*)mxGetData(prhs[2]);
	double *carrierFreq = (double*)mxGetData(prhs[3]);
	double *rot = (double*)mxGetData(prhs[4]);
	
	const size_t *dimF = mxGetDimensions(prhs[3]);
	bool varyingCarrier = dimF[0] > 1 || dimF[1] > 1;

	const size_t *dim = mxGetDimensions(prhs[0]);
	int numDim = mxGetNumberOfDimensions(prhs[0]);
	int N = numDim == 2 ? 1 : dim[2];
	
	if (!mxIsSingle(prhs[0]) )
	{
		mexPrintf("Currently supporting only single class variables of size 64x64 \n");
		return;
	}
	
	const size_t output_dim[3] = {768,128,N};

	plhs[0] = mxCreateNumericArray(3, output_dim, mxUINT8_CLASS, mxREAL);
	unsigned char  *out  = (unsigned char *)mxGetData(plhs[0]);

	int patternSizeX = dim[1];
	int patternSizeY = dim[0];
	//|| dim[0] != 64 || dim[1] != 64

	size_t total_mem_size_phases = dim[0]*dim[1]*N * sizeof(float);
	size_t desired_outputSize = size_t(128*768)*N * sizeof(unsigned char);

	size_t availMemory;
	if (!initCuda(deviceProp, availMemory, total_mem_size_phases+desired_outputSize))
		return;

	size_t max_planesInMemory = 14000; // more causes failues due to number of blocks in the grid(!!!)
	int numIterations = ceil((double)N/max_planesInMemory);

	size_t input_phases_in_memory = min(dim[0]*dim[1]*N * sizeof(float),
										dim[0]*dim[1]*max_planesInMemory * sizeof(float));


	size_t mem_size_out = min(	max_planesInMemory*(768*128), desired_outputSize);

	float *f_freq = new float[N];
	float *f_rot = new float[N];
	for (int k=0;k<N;k++)
	{
		f_freq[k] = varyingCarrier ? carrierFreq[k] : carrierFreq[0];
		f_rot[k] =  varyingCarrier ? rot[k] : rot[0];
	}


	float *d_phases;
	float *d_freq;
	float *d_rot;

	unsigned char *d_out;
	checkCudaErrors(hipMalloc((void **)&d_freq, N*sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&d_rot, N*sizeof(float)));
	checkCudaErrors(hipMemcpy(d_freq, f_freq, N*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_rot, f_rot, N*sizeof(float), hipMemcpyHostToDevice));
	delete f_freq;
	delete f_rot;

	checkCudaErrors(hipMalloc((void **)&d_phases, input_phases_in_memory));
	checkCudaErrors(hipMalloc((void **)&d_out, mem_size_out));
	checkCudaErrors(hipMemset(d_out, 0, mem_size_out));


	int blockSize = 32;
	
	for (int iteration=0;iteration<numIterations;iteration++)
	{
		int startPlane = max_planesInMemory*iteration;
		int endPlane = min((iteration + 1)*max_planesInMemory - 1, N - 1);
		int numPlanes = endPlane-startPlane+1;
		size_t bytesToCopy =  size_t(768*128)*numPlanes;

		size_t input_offset = patternSizeX*patternSizeY*startPlane;
		size_t numBytesOfPhasePatternsToCopy = patternSizeX*patternSizeY*sizeof(float)*numPlanes;
		checkCudaErrors(hipMemcpy(d_phases, phases + input_offset, numBytesOfPhasePatternsToCopy, hipMemcpyHostToDevice));



		// think about the input phases as a concatation of matrices along the column direction.
		// Then, the analysis is done by computing how many block are nedded.
		int numBlocksX = numPlanes*128 / blockSize;
		int numBlocksY = 768 / blockSize;
		
		dim3 dimGrid(numBlocksX, numBlocksY);
		dim3 dimBlock(blockSize, blockSize);
		LeeKernel3 << <dimGrid, dimBlock >> >(d_phases, d_out, numReferencePixels, leeBlockSize, d_freq, patternSizeX, patternSizeY,d_rot);
		checkCudaErrors(hipDeviceSynchronize());

		size_t bytesToCopyOut =  size_t(768*128)*numPlanes;
		size_t out_offset = size_t(startPlane) * size_t(768 * 128);
		checkCudaErrors(hipMemcpy(out+out_offset, d_out, bytesToCopyOut, hipMemcpyDeviceToHost));
	}
	
	// Destroy the handle
	checkCudaErrors(hipFree(d_phases));
	checkCudaErrors(hipFree(d_out));

	checkCudaErrors(hipFree(d_freq));
	checkCudaErrors(hipFree(d_rot));

	hipDeviceReset();
	return;
}
