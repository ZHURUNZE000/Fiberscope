#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "mex.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define MIN(a,b) (a)<(b)?(a):(b)
#define M_PI 3.14159265358979323846

#define DMDwidth  1024
#define DMDheight 768
#define effectiveDMDwidth  DMDheight

__global__ void cudaKernel(bool *dev_binaryPatterns, double *dev_carrierWave, double *dev_inputPhases, int *patternSizeX, int *patternSizeY, int *numReferencePixels, int *leeBlockSize)
{
	int pattern = threadIdx.x;
 
	long long output_offset = DMDwidth*DMDheight*pattern;
	long long input_offset = (*patternSizeX) * (*patternSizeY)*pattern;
	double phaseAngle = 0;


	for (int x = 0; x < DMDwidth; x++)
	{
		int sampleX = (x - (*numReferencePixels)) / (*leeBlockSize);
		for (int y = 0; y < DMDheight; y++)
		{

			phaseAngle = 0.0;

			if (y >= (*numReferencePixels) && y < DMDheight - (*numReferencePixels) && x >= (*numReferencePixels) && x < effectiveDMDwidth - (*numReferencePixels))
			{
				int sampleY = (y - (*numReferencePixels)) / (*leeBlockSize);
				assert(sampleX >= 0 && sampleY >= 0 && sampleX < patternSizeX && sampleY < patternSizeY);
				phaseAngle = dev_inputPhases[input_offset + sampleX*(*patternSizeY) + sampleY];
			}
			dev_binaryPatterns[output_offset + x*DMDheight + y] = (0.5 * (1 + cos(dev_carrierWave[x*DMDheight + y] - phaseAngle))) > 0.5;
		}
	}

}



void mexFunction(int nlhs, mxArray *plhs[],
	int nrhs, const mxArray *prhs[]) {
	hipError_t cudaStatus;

	if (nrhs < 3 || nlhs != 1)
	{
		mexPrintf("Use: OutputBinaryPatterns = CudaProject(inputPhases (NxNxM), numReferencePixels, leeBlockSize, selectedCarrier);");
		return;
	}

	double *inputPhases = (double*)mxGetData(prhs[0]);
	int numReferencePixels = *(double*)mxGetData(prhs[1]);
	int leeBlockSize = *(double*)mxGetData(prhs[2]);
	double selectedCarrier = *(double*)mxGetData(prhs[3]);

	const int numDim = mxGetNumberOfDimensions(prhs[0]);
	const size_t *dataSize = mxGetDimensions(prhs[0]);
	int numPatterns = 1;
	int patternSizeX = dataSize[0];
	int patternSizeY = dataSize[1];
	if (numDim > 2)
	{
		numPatterns = dataSize[2];
	}

	// allocate memory for output on host computer
	const size_t outputDimSize[3] = { DMDheight, DMDwidth, numPatterns };
	plhs[0] = mxCreateLogicalArray(3, outputDimSize);
	bool* binaryPatterns = (bool*)mxGetData(plhs[0]);
	double *carrierWave = new double[DMDwidth*DMDheight];
	for (int x = 0; x < DMDwidth; x++)
	{
		for (int y = 0; y < DMDheight; y++)
		{
			carrierWave[x*DMDheight + y] = 2.0 * M_PI*(x - y)*selectedCarrier;
		}
	}
	// allocate memory on GPU
	double *dev_carrierWave = 0;
	bool *dev_binaryPatterns = 0;
	double *dev_inputPhases = 0;
	int *dev_patternSizeX = 0;
	int *dev_patternSizeY = 0;
	int *dev_numReferencePixels = 0;
	int *dev_leeBlockSize = 0;

	cudaStatus = hipMalloc((void**)&dev_patternSizeX,  sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_patternSizeY, sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_numReferencePixels, sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_leeBlockSize, sizeof(int));


	cudaStatus = hipMalloc((void**)&dev_carrierWave, DMDheight*DMDwidth * sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_binaryPatterns, DMDheight*DMDwidth * numPatterns* sizeof(bool));
	if (cudaStatus != hipSuccess) {
		mexPrintf( "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_inputPhases, patternSizeX*patternSizeY * numPatterns* sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_carrierWave, carrierWave, DMDheight*DMDwidth * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_inputPhases, inputPhases, patternSizeX*patternSizeY * numPatterns* sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMemcpy failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(dev_patternSizeX, &patternSizeX, sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_patternSizeY, &patternSizeY, sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_leeBlockSize, &leeBlockSize, sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_numReferencePixels, &numReferencePixels, sizeof(int), hipMemcpyHostToDevice);


	// Launch a kernel on the GPU with one thread for each element.
	cudaKernel <<< 1, numPatterns >>>(dev_binaryPatterns, dev_carrierWave, dev_inputPhases, dev_patternSizeX, dev_patternSizeY, dev_numReferencePixels, dev_leeBlockSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		mexPrintf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(binaryPatterns, dev_binaryPatterns, DMDheight*DMDwidth * numPatterns* sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipMemcpy failed!");
		goto Error;
	}


	/*
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
		return;
    }
	*/
	delete carrierWave;
Error:
	hipFree(dev_carrierWave);
	hipFree(dev_binaryPatterns);
	hipFree(dev_inputPhases);

	hipFree(dev_patternSizeX);
	hipFree(dev_patternSizeY);
	hipFree(dev_numReferencePixels);
	hipFree(dev_leeBlockSize);


	return;
}
